
#include <hip/hip_runtime.h>
#include <limits>
#include <chrono>
#include <random>
#include <algorithm>
#include <iostream>
#include <iomanip>

constexpr int MAX_CITIES = 15;

template <class T>
struct counting_iterator {

private:
  typedef counting_iterator<T> self;

public:
  typedef T value_type;
  typedef typename std::make_signed<T>::type difference_type;
  typedef T const* pointer;
  typedef T const& reference;
  typedef std::random_access_iterator_tag iterator_category;

  explicit counting_iterator(value_type v) : value(v) { }

  value_type operator*() const { return value; }
  value_type operator[](difference_type n) const { return value + n; }

  self& operator++() { ++value; return *this; }
  self operator++(int) {
    self result{value};
    ++value;
    return result;
  }
  self& operator--() { --value; return *this; }
  self operator--(int) {
    self result{value};
    --value;
    return result;
  }
  self& operator+=(difference_type n) { value += n; return *this; }
  self& operator-=(difference_type n) { value -= n; return *this; }

  friend self operator+(self const& i, difference_type n) {
    return self(i.value + n);
  }
  friend self operator+(difference_type n, self const& i) {
    return self(i.value + n);
  }
  friend difference_type operator-(self const& x, self const& y) {
    return x.value - y.value;
  }
  friend self operator-(self const& i, difference_type n) {
    return self(i.value - n);
  }

  friend bool operator==(self const& x, self const& y) {
    return x.value == y.value;
  }
  friend bool operator!=(self const& x, self const& y) {
    return x.value != y.value;
  }
  friend bool operator<(self const& x, self const& y) {
    return x.value < y.value;
  }
  friend bool operator<=(self const& x, self const& y) {
    return x.value <= y.value;
  }
  friend bool operator>(self const& x, self const& y) {
    return x.value > y.value;
  }
  friend bool operator>=(self const& x, self const& y) {
    return x.value >= y.value;
  }
private:
  value_type value;
};

template <class T,
          class = typename std::enable_if<std::is_integral<T>::value>::type>
inline counting_iterator<T> make_counter(T value) {
  return counting_iterator<T>{value};
}

char const* city_names[MAX_CITIES] = {
  "Atlanta", "Baltimore", "Cleveland", "Denver", "El Paso", "Fort Collins",
  "Green Bay", "Houston", "Indianapolis", "Jacksonville", "Knoxville",
  "Los Angeles", "Memphis", "Nashville", "Orlando"
};

int* init(int N) {
  int* distances = new int[N * N];
  std::mt19937 r;
  std::shuffle(city_names, city_names + N, r);
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      if (i == j) {
        distances[i*N + j] = 9999;
      } else if (city_names[i][0] + 1 == city_names[j][0]) {
        distances[i*N + j] = (r() % 15) + 5;
      } else {
        distances[i*N + j] = (r() % 900) + 100;
      }
    }
  }
  return distances;
}

struct route_cost {
  long route;
  int cost;
  __host__ __device__ route_cost() : route(-1), cost(1 << 30) { }
  __host__ __device__ route_cost(long route, int cost)
    : route(route), cost(cost) { }
  static struct min_class {
    route_cost operator()(route_cost const& x, route_cost const& y) const {
      return x.cost < y.cost ? x : y;
    }
  } min;
  static __host__ __device__ route_cost minf(
      route_cost const& x, route_cost const& y) {
    return x.cost < y.cost ? x : y;
  }
};
route_cost::min_class route_cost::min;

struct route_iterator {
  long remainder;
  int hops_left;
  unsigned visited = 0;
  __host__ __device__ route_iterator(long route_id, int num_hops)
    : remainder(route_id), hops_left(num_hops)
  { }
  __host__ __device__ bool done() const {
    return hops_left <= 0;
  }
  __host__ __device__ int first() {
    int index = (int)(remainder % hops_left);
    remainder /= hops_left;
    --hops_left;
    visited = (1 << index);
    return index;
  }
  __host__ __device__ int next() {
    long available = remainder % hops_left;
    remainder /= hops_left;
    --hops_left;
    int index = 0;
    while (true) {
      if ((visited & (1 << index)) == 0) {
        if (--available < 0) {
          break;
        }
      }
      ++index;
    }
    visited |= (1 << index);
    return index;
  }
};

long factorial(long x) {
  if (x <= 1) {
    return 1;
  }
  return x * factorial(x - 1);
}

__global__ void find_best_kernel(int* distances, int N, long num_routes,
                                 route_cost* block_best) {
  static __shared__ long warp_best_id[32];
  static __shared__ int warp_best_cost[32];
  route_cost local_best;
  for (long i = blockIdx.x * blockDim.x + threadIdx.x; i < num_routes;
       i += blockDim.x * gridDim.x) {
    int cost = 0;
    route_iterator it(i, N);
    int from = it.first();
    while (!it.done()) {
      int to = it.next();
      cost += distances[from*N + to];
      from = to;
    }
    local_best = route_cost::minf(local_best, route_cost(i, cost));
  }
  int lane = threadIdx.x % warpSize;
  int warpId = threadIdx.x / warpSize;
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    local_best = route_cost::minf(local_best, route_cost(
          __shfl_down_sync(~0u, local_best.route, offset),
          __shfl_down_sync(~0u, local_best.cost, offset)));
  }
  if (lane == 0) {
    warp_best_id[warpId] = local_best.route;
    warp_best_cost[warpId] = local_best.cost;
  }
  __syncthreads();
  if (warpId == 0) {
    local_best = route_cost(warp_best_id[lane], warp_best_cost[lane]);
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      local_best = route_cost::minf(local_best, route_cost(
            __shfl_down_sync(~0u, local_best.route, offset),
            __shfl_down_sync(~0u, local_best.cost, offset)));
    }
    if (lane == 0) {
      block_best[blockIdx.x] = local_best;
    }
  }
}

route_cost find_best_route(int const* distances, int N) {
  int* dev_distances;
  hipMalloc(&dev_distances, N * N * sizeof(int));
  hipMemcpy(dev_distances, distances, N * N * sizeof(float),
             hipMemcpyHostToDevice);
  long num_routes = factorial(N);
  int threads = 1024;
  int blocks = std::min((num_routes + threads - 1) / threads, 1024L);
  route_cost* block_best;
  hipMalloc(&block_best, blocks * sizeof(route_cost));
  find_best_kernel<<<blocks, threads>>>(dev_distances, N, num_routes,
                                        block_best);
  hipDeviceSynchronize();
  route_cost* host_block_best = new route_cost[blocks];
  hipMemcpy(host_block_best, block_best, blocks * sizeof(route_cost),
             hipMemcpyDeviceToHost);
  route_cost best_route;
  for (int i = 0; i < blocks; ++i) {
    best_route = route_cost::minf(best_route, host_block_best[i]);
  }
  hipFree(block_best);
  hipFree(dev_distances);
  delete[] host_block_best;
  return best_route;
}

void print_route(route_cost best_route, int N) {
  std::cout << "Best route: " << best_route.cost << " miles\n";
  route_iterator it(best_route.route, N);
  std::cout << city_names[it.first()];
  while (!it.done()) {
    std::cout << ", " << city_names[it.next()];
  }
  std::cout << "\n";
}

int main(int argc, char **argv) {
  int N = argc < 2 ? 5 : std::atoi(argv[1]);
  if (N < 1 || N > MAX_CITIES) {
    std::cout << N << " must be between 1 and " << MAX_CITIES << ".\n";
    return 1;
  }
  int const* distances = init(N);

  find_best_route(distances, std::min(N, 5));

  std::cout << "Checking " << factorial(N) 
            << " routes for the best way to visit " << N << " cities...\n";
  auto start = std::chrono::steady_clock::now();

  auto best_route = find_best_route(distances, N);

  auto end = std::chrono::steady_clock::now();
  auto duration =
    std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
  std::cout << "Took " << (duration / 1000) << "." << std::setw(3) 
            << std::setfill('0') << (duration % 1000) << "s\n";

  print_route(best_route, N);
}
